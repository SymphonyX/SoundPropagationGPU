#include "MarshalStructs.h"

extern "C" void runMainLoopKernel(int columns, int rows, SoundGridStruct* soundMap, SoundSourceStruct* soundSource) 
{
	dim3 blocks(1,1);
	dim3 threads(columns, rows);

	
	SoundGridStruct* soundMap_dev;
	hipMalloc((void**)&soundMap_dev, (rows*columns)*sizeof(SoundGridStruct));
	hipMemcpy(soundMap_dev, soundMap, (rows*columns)*sizeof(SoundGridStruct), hipMemcpyHostToDevice);

	SoundSourceStruct* soundSource_dev;
	hipMemcpy(soundSource_dev, soundSource, sizeof(SoundSourceStruct), hipMemcpyHostToDevice);

	//cudaEmit()
	//cudaMerge()
	//cudaScatter()
	//cudaCollect()
	//cudaTick()
}