#include "hip/hip_runtime.h"
#include "MarshalStructs.cu"
#include <math.h>

#define NUMBER_OF_DIRECTIONS 4

enum {North = 0, East = 1, South = 2, West = 3, None = -1} Direction;

//***********HELPER FUNCTIONS*************//

__device__ int reverseDirection(int direction)
{
	if (direction == West) return East;
	else if (direction == East) return West;
	else if (direction == North) return South;
	else if (direction == South) return North;
	else return None;
}

__device__ int clockwiseDirection(int direction)
{
	if (direction == West) return North;
	else if (direction == East) return South;
	else if (direction == North) return East;
	else if (direction == South) return West;
	else return None;
}

__device__ int counterClockwiseDirection(int direction)
{
	if (direction == West) return South;
	else if (direction == East) return North;
	else if (direction == North) return West;
	else if (direction == South) return East;
	else return None;
}

__device__ SoundGridStruct* neighborAtDirection(SoundGridStruct* soundMap, SoundGridStruct* soundGrid, int direction, int rows, int columns)
{
	if (direction == North && soundGrid->z > 0) {return &soundMap[(soundGrid->z-1)*columns+soundGrid->x]; }
	else if (direction == East && soundGrid->x < columns) {return &soundMap[soundGrid->z*columns+(soundGrid->x+1)]; }
	else if (direction == West && soundGrid->x > 0) {return &soundMap[soundGrid->z*columns+(soundGrid->x-1)]; }
	else if (direction == South && soundGrid->z < rows) {return &soundMap[(soundGrid->z+1)*columns+soundGrid->x]; }
	else return NULL;
}

//************KERNELS******************//

__global__ void emitKernel(SoundSourceStruct* soundSource, SoundGridStruct* soundMap, int rows, int columns, int tick)
{
	int x = threadIdx.x;
	int y = blockIdx.x;

	tick = tick % 150;

	if (soundSource->x == x && soundSource->z == y)
	{
		SoundGridStruct* soundGrid = &soundMap[y*columns+x];
		int frameSize = soundSource->sizesOfPacketList[tick];
		for (int index = 0; index < frameSize; index++) 
		{
			for (int direction = 0; direction < NUMBER_OF_DIRECTIONS; direction++)
			{
				int nextIndex = soundGrid->sizeOfIn[direction];
				soundGrid->IN[direction][nextIndex] = soundSource->packetList[tick][index];
				soundGrid->sizeOfIn[direction] = nextIndex+1;
			}
		}

	}
}

__global__ void mergeKernel(SoundGridStruct* soundMap, int rows, int columns)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	SoundGridStruct* soundGrid = &soundMap[y*columns+x];
	soundGrid->updated = false;
	for (int direction = 0; direction < NUMBER_OF_DIRECTIONS; direction++)
	{
		SoundPacketStruct soundPacket = SoundPacketStruct(0.0f);
		for (int i = 0; i < soundGrid->sizeOfIn[direction]; i++)
		{
			SoundPacketStruct* packetListPtr = soundGrid->IN[direction]; 
			soundPacket.amplitude += (packetListPtr+i)->amplitude;
		}
		soundGrid->sizeOfIn[direction] = 0;

		if (abs(soundPacket.amplitude) > soundGrid->epsilon)
		{
			SoundPacketStruct* packetListPtr = soundGrid->IN[direction];
			*(packetListPtr+soundGrid->sizeOfIn[direction]) = soundPacket;
		}
		soundGrid->updated = true;
	}
}

__global__ void scatterKernel(SoundGridStruct* soundMap, int rows, int columns)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	SoundGridStruct* soundGrid = &soundMap[y*columns+x];
	soundGrid->updated = false;
	
	for (int direction = 0; direction < NUMBER_OF_DIRECTIONS; direction++)
	{
		soundGrid->sizeOfOut[direction] = 0;
	}

	for (int direction = 0; direction < NUMBER_OF_DIRECTIONS; direction++)
	{
		SoundPacketStruct* inVector = soundGrid->IN[direction];
		if (soundGrid->flagWall)
		{
			SoundPacketStruct* outVector = soundGrid->OUT[direction];
			for (int index = 0; index < soundGrid->sizeOfIn[direction]; index++)
			{
				SoundPacketStruct* packet = &inVector[index];
				if (abs(packet->amplitude > soundGrid->epsilon))
				{
					SoundPacketStruct packetCopy = SoundPacketStruct(packet->amplitude * soundGrid->reflectionRate);
					packetCopy.maxRange = packet->maxRange;
					packetCopy.minRange = packet->minRange;
					*(outVector+soundGrid->sizeOfOut[direction]) = packetCopy;
					soundGrid->sizeOfOut[direction] += 1;
					soundGrid->updated = true;
				}

			}
		} 
		else 
		{
			SoundPacketStruct* forwardVector = soundGrid->OUT[reverseDirection(direction)];
			SoundPacketStruct* backwardVector = soundGrid->OUT[direction];
			SoundPacketStruct* clockwiseVector = soundGrid->OUT[clockwiseDirection(direction)];
			SoundPacketStruct* counterClockwiseVector = soundGrid->OUT[counterClockwiseDirection(direction)];

			for (int index = 0; index < soundGrid->sizeOfIn[direction]; index++)
			{
				SoundPacketStruct* soundPacket = (inVector+index);
				if (abs(soundPacket->amplitude > soundGrid->epsilon))
				{
					int* fwdVectorSize = &soundGrid->sizeOfOut[reverseDirection(direction)];
					*(forwardVector+*fwdVectorSize) = SoundPacketStruct(soundGrid->absorptionRate * soundPacket->amplitude / 2, soundPacket->minRange, soundPacket->maxRange);
					*fwdVectorSize +=1;

					int* backwardVectorSize = &soundGrid->sizeOfOut[direction];
					*(backwardVector+*backwardVectorSize) = SoundPacketStruct(soundGrid->absorptionRate * -soundPacket->amplitude / 2, soundPacket->minRange, soundPacket->maxRange);
					*backwardVectorSize +=1;

					int* clockwiseVectorSize = &soundGrid->sizeOfOut[clockwiseDirection(direction)];
					*(clockwiseVector+*clockwiseVectorSize) = SoundPacketStruct(soundGrid->absorptionRate * soundPacket->amplitude / 2, soundPacket->minRange, soundPacket->maxRange);
					*clockwiseVectorSize +=1;

					int* counterClockwiseVectorSize = &soundGrid->sizeOfOut[counterClockwiseDirection(direction)];
					*(counterClockwiseVector+*counterClockwiseVectorSize) = SoundPacketStruct(soundGrid->absorptionRate * soundPacket->amplitude / 2, soundPacket->minRange, soundPacket->maxRange);
					*counterClockwiseVectorSize +=1;

					soundGrid->updated = true;
				}
			}
		}
	}
}

__global__ void collectKernel(SoundGridStruct* soundMap, int rows, int columns)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	SoundGridStruct* soundGrid = &soundMap[y*columns+x];
	
	for (int direction = 0; direction < NUMBER_OF_DIRECTIONS; direction++)
	{
		int revDirection = reverseDirection(direction);
		SoundPacketStruct* frame = soundGrid->OUT[direction];
		SoundGridStruct* neighbor = neighborAtDirection(soundMap, soundGrid, direction, rows, columns);
		SoundPacketStruct* neighborFrame = NULL;
		if (neighbor != NULL)
		{
			neighborFrame = neighbor->IN[revDirection];
		}

		if (neighborFrame != NULL)
		{
			neighbor->sizeOfIn[revDirection] = 0;
			for (int index = 0; index < soundGrid->sizeOfOut[direction]; index++)
			{
				SoundPacketStruct packet = SoundPacketStruct(frame[index].amplitude, frame[index].minRange, frame[index].maxRange); 
				neighborFrame[index] = packet;
				neighbor->sizeOfIn[revDirection] += 1;
			}
		}
	}
}

extern "C" void runMainLoopKernel(int columns, int rows, SoundGridStruct* soundMap, SoundSourceStruct* soundSource, int tick) 
{
	dim3 blocks(64, 1, 1);
	dim3 threads(64, 1, 1);

	
	SoundGridStruct* soundMap_dev;
	hipMalloc((void**)&soundMap_dev, (rows*columns)*sizeof(SoundGridStruct));
	hipMemcpy(soundMap_dev, soundMap, (rows*columns)*sizeof(SoundGridStruct), hipMemcpyHostToDevice);

	SoundSourceStruct* soundSource_dev;
	hipMalloc((void**)&soundSource_dev, sizeof(SoundSourceStruct));
	hipMemcpy(soundSource_dev, soundSource, sizeof(SoundSourceStruct), hipMemcpyHostToDevice);

	emitKernel<<<blocks, threads>>> (soundSource_dev, soundMap_dev, rows, columns, tick);
	//mergeKernel<<<blocks, threads>>> (soundMap_dev, rows, columns);
	//scatterKernel<<<blocks, threads>>> (soundMap_dev, rows, columns);
	//collectKernel<<<blocks, threads>>> (soundMap_dev, rows, columns);

	hipMemcpy(soundMap, soundMap_dev, (rows*columns)*sizeof(SoundGridStruct), hipMemcpyDeviceToHost);
	hipMemcpy(soundSource, soundSource_dev, sizeof(SoundSourceStruct), hipMemcpyDeviceToHost);
}